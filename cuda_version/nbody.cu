#include "nbody.h"

// nvcc -o o nbody.cu

int main() {
    // Initialize particle size and duration
    int num_particles = 10000;
    const int num_steps = 10000;
    const int block_size = 256;
    const int num_blocks = (num_particles + block_size - 1) / block_size;
    printf("block size: %d and num blocks: %d\n", block_size, num_blocks);
    
    // Generate n particles
    Particle* particles = generate_random_particles(&num_particles,x_limit,y_limit,G);

    float start_cpu, finish_cpu, elapsed_cpu, elapsedTime;
    hipEvent_t start, stop;

    // start time
    GET_TIME(start_cpu);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    // Simulate force interactions
    for (int time_step = 1; time_step < num_steps; time_step++) {

        // Create root node
        Node* root = create_node(x_limit/2, y_limit/2, 1, x_limit/2);
        for (int i = 0; i < num_particles; i++) {
            insert(root, &particles[i]);
        }

        Particle* d_particles;
        hipMalloc(&d_particles, num_particles*sizeof(Particle));
        hipMemcpy(d_particles, particles, num_particles*sizeof(Particle), hipMemcpyHostToDevice);
        
        Node* d_root;
        hipMalloc(&d_root, sizeof(Node));
        hipMemcpy(d_root, root, sizeof(Node), hipMemcpyHostToDevice);

        // calculate forces in GPU
        calculate_force_kernel<<<num_blocks, block_size>>>(d_particles, d_root, num_particles);
        update_positions_kernel<<<num_blocks, block_size>>>(d_particles, time_step, num_particles);
        
        // particle_print_gpu<<<num_blocks, block_size>>>(d_particles, num_particles);
        // printf("----\n");

        hipMemcpy(root, d_root, sizeof(Node), hipMemcpyDeviceToHost);
        hipMemcpy(particles, d_particles, num_particles * sizeof(Particle), hipMemcpyDeviceToHost);

        hipFree(d_particles);
        hipFree(d_root);
        free_tree(root);
        root = NULL;
    }

    // end time
    hipDeviceSynchronize();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Time to generate:  %e s\n", elapsedTime/1000);
    GET_TIME(finish_cpu);
    elapsed_cpu = finish_cpu - start_cpu;
    printf("Elapsed time = %e seconds\n", elapsed_cpu);
    
    // Free memory for particles
    free(particles);

    particles = NULL;
    return 0;
}
